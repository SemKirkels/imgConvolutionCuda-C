#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define NUMBER_OF_IMAGES 10

typedef struct Pixel
{
    unsigned char r, g, b, a;
} Pixel;

__global__ void ConvertImageToGrayCpu(unsigned char *originalImage, unsigned char *imageDataGrayscale, int width, int height);
__global__ void convolveImage(unsigned char *imageDataGrayscale, unsigned char *imageDataConvolution, int width, int height);
__global__ void minPooling(unsigned char *originalImage, unsigned char *minPoolingImage, int width, int height);
__global__ void maxPooling(unsigned char *originalImage, unsigned char *maxPoolingImage, int width, int height);

int main(int argc, char **argv)
{
    clock_t timer_start, timer_end, timer_start_process, timer_end_process;
    timer_start = clock();

    size_t threadsPerBlock = 128;
    size_t numberOfBlocks = 32;

    hipStream_t stream[10];

    for (int i = 0; i < 10; i++)
    {
        hipStreamCreate(&stream[i]);
    }

    printf("Building filepaths\r\n");

    const char *inputFileName[10] =
        {
            "Images/img_0.png",
            "Images/img_1.png",
            "Images/img_2.png",
            "Images/img_3.png",
            "Images/img_4.png",
            "Images/img_5.png",
            "Images/img_6.png",
            "Images/img_7.png",
            "Images/img_8.png",
            "Images/img_9.png",
        };

    // Build output filename
    const char *fileNameOutConvolution[10] =
        {
            "Output_Images/Convolution/OutputConvolution0.png",
            "Output_Images/Convolution/OutputConvolution1.png",
            "Output_Images/Convolution/OutputConvolution2.png",
            "Output_Images/Convolution/OutputConvolution3.png",
            "Output_Images/Convolution/OutputConvolution4.png",
            "Output_Images/Convolution/OutputConvolution5.png",
            "Output_Images/Convolution/OutputConvolution6.png",
            "Output_Images/Convolution/OutputConvolution7.png",
            "Output_Images/Convolution/OutputConvolution8.png",
            "Output_Images/Convolution/OutputConvolution9.png",
        };

    const char *fileNameOutMinPooling[10] =
        {
            "Output_Images/Pooling/OutputMinPooling0.png",
            "Output_Images/Pooling/OutputMinPooling1.png",
            "Output_Images/Pooling/OutputMinPooling2.png",
            "Output_Images/Pooling/OutputMinPooling3.png",
            "Output_Images/Pooling/OutputMinPooling4.png",
            "Output_Images/Pooling/OutputMinPooling5.png",
            "Output_Images/Pooling/OutputMinPooling6.png",
            "Output_Images/Pooling/OutputMinPooling7.png",
            "Output_Images/Pooling/OutputMinPooling8.png",
            "Output_Images/Pooling/OutputMinPooling9.png",
        };

    const char *fileNameOutMaxPooling[10] =
        {
            "Output_Images/Pooling/OutputMaxPooling0.png",
            "Output_Images/Pooling/OutputMaxPooling1.png",
            "Output_Images/Pooling/OutputMaxPooling2.png",
            "Output_Images/Pooling/OutputMaxPooling3.png",
            "Output_Images/Pooling/OutputMaxPooling4.png",
            "Output_Images/Pooling/OutputMaxPooling5.png",
            "Output_Images/Pooling/OutputMaxPooling6.png",
            "Output_Images/Pooling/OutputMaxPooling7.png",
            "Output_Images/Pooling/OutputMaxPooling8.png",
            "Output_Images/Pooling/OutputMaxPooling9.png",
        };

    int width[10], height[10], componentCount[10], size[10];

    unsigned char *originalImageHost[10];
    unsigned char *imageDataConvolutionHost[10]; // Saves output image
    unsigned char *imageDataMinPoolingHost[10];  // Saves Min pooling image
    unsigned char *imageDataMaxPoolingHost[10];  // Saves Max pooling image
    unsigned char *originalImage[10];            // Saves the original image on host
    unsigned char *imageDataGrayscale[10];       // Saves the grayscale image on device
    unsigned char *imageDataConvolution[10];     // Saves the convolved image
    unsigned char *imageDataMinPooling[10];      // Saves the min pooled image
    unsigned char *imageDataMaxPooling[10];      // Saves the max pooled image

    printf("Done\r\n");

    printf("Loading png files\r\n");

    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        originalImageHost[i] = stbi_load(inputFileName[i], &width[i], &height[i], &componentCount[i], 4);

        size[i] = height[i] * width[i] * 4;

        // Saves output image
        imageDataConvolutionHost[i] = (unsigned char *)malloc(size[i]);

        // Saves Min pooling image
        imageDataMinPoolingHost[i] = (unsigned char *)malloc(size[i]);

        // Saves Max pooling image
        imageDataMaxPoolingHost[i] = (unsigned char *)malloc(size[i]);

        hipMalloc(&originalImage[i], size[i]);
        hipMalloc(&imageDataGrayscale[i], size[i]);
        hipMalloc(&imageDataConvolution[i], size[i]);
        hipMalloc(&imageDataMinPooling[i], size[i]);
        hipMalloc(&imageDataMaxPooling[i], size[i]);

        hipMemcpy(originalImage[i], originalImageHost[i], size[i], hipMemcpyHostToDevice);
    }

    printf("Done\r\n");

    timer_start_process = clock();

    // Process grayscale
    printf("Processing images grayscale\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        ConvertImageToGrayCpu<<<numberOfBlocks, threadsPerBlock, i, stream[i]>>>(originalImage[i], imageDataGrayscale[i], width[i], height[i]);
    }
    hipDeviceSynchronize();
    printf("Done\r\n");

    // Process convolution
    printf("Processing image convolution\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        convolveImage<<<numberOfBlocks, threadsPerBlock, i, stream[i]>>>(imageDataGrayscale[i], imageDataConvolution[i], width[i], height[i]);
    }
    hipDeviceSynchronize();
    printf("Done\r\n");

    // Process min pooling
    printf("Processing images minimum pooling\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        dim3 gridDim(width[i] / 2, height[i] / 2);
        dim3 blockDim(2, 2);
        // minPooling<<<numberOfBlocks, threadsPerBlock, i, stream[i]>>>(originalImage[i], imageDataMinPooling[i], width[i], height[i]);
        minPooling<<<gridDim, blockDim, i, stream[i]>>>(originalImage[i], imageDataMinPooling[i], width[i], height[i]);
    }
    hipDeviceSynchronize();
    printf("Done\r\n");

    // Process max pooling
    printf("Processing image maximum pooling\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        dim3 gridDim(width[i] / 2, height[i] / 2);
        dim3 blockDim(2, 2);
        // maxPooling<<<numberOfBlocks, threadsPerBlock, i, stream[i]>>>(originalImage[i], imageDataMaxPooling[i], width[i], height[i]);
        maxPooling<<<gridDim, blockDim, i, stream[i]>>>(originalImage[i], imageDataMaxPooling[i], width[i], height[i]);
    }
    hipDeviceSynchronize();
    printf("Done\r\n");

    timer_end_process = clock();

    // Writing Convolved images
    printf("Writing convolved png to disk\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        printf("Image %d of %d\r\n", i + 1, 10);
        hipMemcpy(imageDataConvolutionHost[i], imageDataConvolution[i], size[i], hipMemcpyDeviceToHost);
        stbi_write_png(fileNameOutConvolution[i], width[i] - 2, height[i] - 2, 4, imageDataConvolutionHost[i], 4 * width[i]);
    }
    printf("Done\r\n");

    // Writing min pooled images
    printf("Writing min pooling png to disk\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        printf("Image %d of %d\r\n", i + 1, 10);
        hipMemcpy(imageDataMinPoolingHost[i], imageDataMinPooling[i], size[i], hipMemcpyDeviceToHost);
        stbi_write_png(fileNameOutMinPooling[i], width[i] / 2, height[i] / 2, 4, imageDataMinPoolingHost[i], 4 * (width[i] / 2));
    }
    printf("Done\r\n");

    // Writing max pooled images
    printf("Writing max pooling png to disk\r\n");
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        printf("Image %d of %d\r\n", i + 1, 10);
        hipMemcpy(imageDataMaxPoolingHost[i], imageDataMaxPooling[i], size[i], hipMemcpyDeviceToHost);
        stbi_write_png(fileNameOutMaxPooling[i], width[i] / 2, height[i] / 2, 4, imageDataMaxPoolingHost[i], 4 * (width[i] / 2));
    }
    printf("Done\r\n");

    // Free memory and destroy streams
    for (int i = 0; i < NUMBER_OF_IMAGES; i++)
    {
        stbi_image_free(originalImageHost[i]);

        free(imageDataConvolutionHost[i]);
        free(imageDataMinPoolingHost[i]);
        free(imageDataMaxPoolingHost[i]);

        hipFree(originalImage[i]);
        hipFree(imageDataConvolution[i]);
        hipFree(imageDataMinPooling[i]);
        hipFree(imageDataMaxPooling[i]);

        hipStreamDestroy(stream[i]);
    }

    timer_end = clock(); // end the timer
    double time_spent = (double)(timer_end - timer_start) / CLOCKS_PER_SEC;
    printf("\nTotal program time: %.3fs\n", time_spent);

    double time_spent_process = (double)(timer_end_process - timer_start_process) / CLOCKS_PER_SEC;
    printf("\nProcessing time (CUDA Kernels): %.3fs\n", time_spent_process);

    return 0;
}

__global__ void ConvertImageToGrayCpu(unsigned char *originalImage, unsigned char *imageDataGrayscale, int width, int height)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    int gridStride = blockDim.x * gridDim.x;
    int totalPixels = width * height * 4;

    for (int x = idx; x < totalPixels; x += gridStride)
    {
        Pixel *ptrPixel = (Pixel *)&imageDataGrayscale[x];
        Pixel *ptrPixelOriginal = (Pixel *)&originalImage[x];
        unsigned char pixelValue = (unsigned char)(ptrPixelOriginal->r * 0.2126f + ptrPixelOriginal->g * 0.7152f + ptrPixelOriginal->b * 0.0722f);
        ptrPixel->r = pixelValue;
        ptrPixel->g = pixelValue;
        ptrPixel->b = pixelValue;
        ptrPixel->a = 255;
    }
}

__global__ void convolveImage(unsigned char *imageDataGrayscale, unsigned char *imageDataConvolution, int width, int height)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    int gridStride = blockDim.x * gridDim.x;
    int totalPixels = width * height * 4;

    int kernel[3][3] =
        {
            {1, 0, -1},
            {1, 0, -1},
            {1, 0, -1}};

    int pixels[3][3] = {0};
    int finalPixel = 0;

    for (int x = idx; x < totalPixels - 2; x += gridStride)
    {
        for (int i = 0; i <= 2; i++)
        {
            Pixel *ptrPixel = (Pixel *)&imageDataGrayscale[x + i * 4];

            pixels[0][i] = ptrPixel->r * kernel[0][i];
        }

        for (int i = 0; i <= 2; i++)
        {
            Pixel *ptrPixel = (Pixel *)&imageDataGrayscale[x + width * 4 + i * 4];

            pixels[1][i] = ptrPixel->r * kernel[1][i];
        }

        for (int i = 0; i <= 2; i++)
        {
            Pixel *ptrPixel = (Pixel *)&imageDataGrayscale[x + (2 * width * 4) + i * 4];

            pixels[2][i] = ptrPixel->r * kernel[2][i];
        }

        finalPixel = (pixels[0][0] + pixels[0][1] + pixels[0][2] + pixels[1][0] + pixels[1][1] + pixels[1][2] + pixels[2][0] + pixels[2][1] + pixels[2][2]) / 9;

        Pixel *ptrPixel = (Pixel *)&imageDataConvolution[x];
        ptrPixel->r = finalPixel;
        ptrPixel->g = finalPixel;
        ptrPixel->b = finalPixel;
        ptrPixel->a = 255;
    }
}

__global__ void minPooling(unsigned char *originalImage, unsigned char *minPoolingImage, int width, int height)
{
    // Calculate the 2D block index
    int blockY = blockIdx.y;
    int blockX = blockIdx.x;

    // Calculate the starting position of the 2x2 block
    int y = blockY * 2;
    int x = blockX * 2;

    // Calculate the index of the current pixel in the 1D arrays
    int indexOut = blockY * width / 2 * 4 + blockX * 4;

    // For each channel, find the minimum value in the 2x2 block
    for (int c = 0; c < 4; c++)
    {
        unsigned char min = 255;
        for (int dy = 0; dy < 2; dy++)
        {
            for (int dx = 0; dx < 2; dx++)
            {
                // Calculate the index of the current pixel in the 1D array
                int index = (y + dy) * width * 4 + (x + dx) * 4 + c;
                unsigned char value = originalImage[index];
                min = (value < min) ? value : min;
            }
        }
        // Store the minimum value in the result array
        minPoolingImage[indexOut + c] = min;
    }
}

__global__ void maxPooling(unsigned char *originalImage, unsigned char *minPoolingImage, int width, int height)
{
    // Calculate the 2D block index
    int blockY = blockIdx.y;
    int blockX = blockIdx.x;

    // Calculate the starting position of the 2x2 block
    int y = blockY * 2;
    int x = blockX * 2;

    // Calculate the index of the current pixel in the 1D arrays
    int indexOut = blockY * width / 2 * 4 + blockX * 4;

    // For each channel, find the minimum value in the 2x2 block
    for (int c = 0; c < 4; c++)
    {
        unsigned char min = 0;
        for (int dy = 0; dy < 2; dy++)
        {
            for (int dx = 0; dx < 2; dx++)
            {
                // Calculate the index of the current pixel in the 1D array
                int index = (y + dy) * width * 4 + (x + dx) * 4 + c;
                unsigned char value = originalImage[index];
                min = (value > min) ? value : min;
            }
        }
        // Store the minimum value in the result array
        minPoolingImage[indexOut + c] = min;
    }
}
